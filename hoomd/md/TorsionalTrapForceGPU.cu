#include "hip/hip_runtime.h"
// Copyright (c) 2009-2022 The Regents of the University of Michigan.
// Part of HOOMD-blue, released under the BSD 3-Clause License.

#include "TorsionalTrapForceGPU.cuh"
#include "hoomd/TextureTools.h"
#include <iostream>
#include <math.h>
#include <sstream>
#include <stdexcept>
#include <vector>

#include <assert.h>

#ifdef SINGLE_PRECISION
#define __scalar2int_rn __float2int_rn
#else
#define __scalar2int_rn __double2int_rn
#endif

/*! \file HarmonicDihedralForceGPU.cu
    \brief Defines GPU kernel code for calculating the harmonic dihedral forces. Used by
   HarmonicDihedralForceComputeGPU.
*/

namespace hoomd
    {
namespace md
    {
namespace kernel
    {

//! GPU implementation of anglDiff
__device__ Scalar gpu_anglDiff(Scalar diff)
    {
    if (diff > M_PI)
        {
        diff -= 2 * M_PI;
        }
    else if (diff <= -M_PI)
        {
        diff += 2 * M_PI;
        }
    return diff;
    }
//! Kernel for calculating harmonic dihedral forces on the GPU
/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos particle positions on the device
    \param d_params Parameters for the angle force
    \param box Box dimensions for periodic boundary condition handling
    \param tlist Dihedral data to use in calculating the forces
    \param dihedral_ABCD List of relative atom positions in the dihedrals
    \param pitch Pitch of 2D dihedral list
    \param n_dihedrals_list List of numbers of dihedrals per atom
*/
__global__ void gpu_compute_torsionaltrap_sin_force_kernel(const unsigned int group_size,const BoxDim box,
                                                       const Scalar4* d_pos,
                                                       Scalar4* d_torque,
                                                       unsigned int* d_rtag,
                                                       unsigned int* d_tag_array1,
                                                       unsigned int* d_tag_array2,
                                                       unsigned int* d_tag_array3,
                                                       unsigned int* d_tag_array4,
                                                       Scalar* d_ref_angles,
                                                       Scalar* d_angles,
                                                       Scalar2* d_oldnew_angles,
                                                       Scalar3* d_ref_vecp,
                                                       Scalar3* d_ref_vecn,
                                                       const Index2D d_oldnew_value,
                                                       const Index2D d_ref_vecp_value,
                                                       const Index2D d_ref_vecn_value,
                                                       const typeval_union* d_group_typeval,
                                                       const Scalar* d_params,
                                                       long unsigned int timestep)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= group_size)
        return;
    unsigned int typval = d_group_typeval[group_idx].type;
    //printf("%u \n", typva);
    Scalar params = __ldg(d_params + typval);

    Scalar K = params;


    unsigned int tagp = d_tag_array1[group_idx];
    unsigned int tagn = d_tag_array2[group_idx];
    unsigned int tagpside = d_tag_array3[group_idx];
    unsigned int tagnside = d_tag_array4[group_idx];
    unsigned int rtagp = d_rtag[tagp];
    unsigned int rtagn = d_rtag[tagn];
    unsigned int rtagpside = d_rtag[tagpside];
    unsigned int rtagnside = d_rtag[tagnside];




    Scalar4 pos_b1 = d_pos[rtagp];
    Scalar4 pos_c1 = d_pos[rtagn];
    Scalar4 pos_a1 = d_pos[rtagpside];
    Scalar4 pos_d1 = d_pos[rtagnside];



    Scalar3 a_poss1 = make_scalar3(pos_a1.x,pos_a1.y,pos_a1.z);
    Scalar3 b_poss1 = make_scalar3(pos_b1.x,pos_b1.y,pos_b1.z);
    Scalar3 c_poss1 = make_scalar3(pos_c1.x,pos_c1.y,pos_c1.z);
    Scalar3 d_poss1 = make_scalar3(pos_d1.x,pos_d1.y,pos_d1.z);



    Scalar3 dab1;
    dab1 = a_poss1 - b_poss1;

    Scalar3 ddc1;
    ddc1 = d_poss1 - c_poss1;

    dab1 = box.minImage(dab1);

    ddc1 = box.minImage(ddc1);

    Scalar dab1mag = fast::sqrt(dot(dab1,dab1));
    Scalar ddc1mag = fast::sqrt(dot(ddc1,ddc1));
    Scalar3 unitddc = make_scalar3(ddc1.x/ddc1mag,ddc1.y/ddc1mag,ddc1.z/ddc1mag);
    Scalar3 unitdab = make_scalar3(dab1.x/dab1mag,dab1.y/dab1mag,dab1.z/dab1mag);

    Scalar3 refvecn =__ldg(d_ref_vecn+d_ref_vecn_value(group_idx, typval));
    Scalar3 refvecp =__ldg(d_ref_vecp+d_ref_vecp_value(group_idx, typval));
    Scalar dotp = dot(refvecp,unitdab) ;
    Scalar dotn = dot(refvecn,unitddc) ;

    Scalar3 crossp;
    Scalar3 crossn;
    Scalar x = unitdab.y*refvecp.z - unitdab.z*refvecp.y;
    Scalar y = unitdab.z*refvecp.x - unitdab.x*refvecp.z;
    Scalar z = unitdab.x*refvecp.y - unitdab.y*refvecp.x;
    crossp = make_scalar3(x,y,z);

    Scalar x1 = unitddc.y*refvecn.z - unitddc.z*refvecn.y;
    Scalar y1 = unitddc.z*refvecn.x - unitddc.x*refvecn.z;
    Scalar z1 = unitddc.x*refvecn.y - unitddc.y*refvecn.x;
    crossn = make_scalar3(x1,y1,z1);

    Scalar angl;
    Scalar diffangl;
    Scalar ref_angl;
    Scalar tmpangl;
    Scalar oldangl;
    Scalar3 torqp;
    Scalar3 torqn;
    Scalar3 constT;
    torqp = make_scalar3(0.0,0.0,0.0);
    torqn = make_scalar3(0.0,0.0,0.0);
    oldangl = 0;
    diffangl = 0;
    ref_angl = 0;
    angl = 0;




    tmpangl = atan2(dab1.y, dab1.x) - atan2(ddc1.y, ddc1.x);
    tmpangl = gpu_anglDiff(tmpangl);
    Scalar2 TMPoldnew_angles = __ldg(d_oldnew_angles+d_oldnew_value(group_idx, typval));
    oldangl = TMPoldnew_angles.x;
    diffangl = tmpangl - oldangl;
    diffangl = gpu_anglDiff(diffangl);
    TMPoldnew_angles.y = tmpangl;

    d_TMP_angles = d_angles[group_idx];
    angl = d_TMP_angles+diffangl;
    ref_angl = d_ref_angles[group_idx];
    //printf("%d %f \n",i,angl);
    d_angles[group_idx] = angl;

    TMPoldnew_angles.x = tmpangl;
    d_oldnew_angles[d_oldnew_value(group_idx, typval)] = TMPoldnew_angles;
    Scalar tmagp = 2*K*dotp;
    Scalar tmagn = 2*K*dotn;

    torqp = make_scalar3(tmagp*crossp.x,tmagp*crossp.y,tmagp*crossp.z);
    torqn = make_scalar3(tmagn*crossn.x,tmagn*crossn.y,tmagn*crossn.z);
    d_torque[rtagp] = make_scalar4(torqp.x,torqp.y,torqp.z,0);
    d_torque[rtagn] = make_scalar4(torqn.x,torqn.y,torqn.z,0);



    }

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos particle positions on the GPU
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param tlist Dihedral data to use in calculating the forces
    \param dihedral_ABCD List of relative atom positions in the dihedrals
    \param pitch Pitch of 2D dihedral list
    \param n_dihedrals_list List of numbers of dihedrals per atom
    \param d_params K, sign,multiplicity params packed as padded Scalar4 variables
    \param n_dihedral_types Number of dihedral types in d_params
    \param block_size Block size to use when performing calculations
    \param compute_capability Compute capability of the device (200, 300, 350, ...)

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one Scalar4 element per dihedral type. The x component contains K the
   spring constant and the y component contains sign, and the z component the multiplicity.
*/
hipError_t gpu_compute_torsionaltrap_sin_forces(const unsigned int group_size,const BoxDim& box,
                                                const Scalar4* d_pos,
                                                Scalar4* d_torque,
                                                unsigned int* d_rtag,
                                                unsigned int* d_tag_array1,
                                                unsigned int* d_tag_array2,
                                                unsigned int* d_tag_array3,
                                                unsigned int* d_tag_array4,
                                                Scalar* d_ref_angles,
                                                Scalar* d_angles,
                                                Scalar2* d_oldnew_angles,
                                                Scalar3* d_ref_vecp,
                                                Scalar3* d_ref_vecn,
                                                const Index2D& d_oldnew_value,
                                                const Index2D& d_ref_vecp_value,
                                                const Index2D& d_ref_vecn_value,
                                                const typeval_union* d_group_typeval,
                                                Scalar* d_params,
                                                long unsigned int timestep,
                                                unsigned int block_size)
                                                //,int warp_size)
    {
    assert(d_params);
    // setup the grid to run the kernel
    dim3 grid(group_size / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);
    // unsigned int max_block_size;
    // hipFuncAttributes attr;
    // hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_compute_harmonic_dihedral_forces_kernel);
    // max_block_size = attr.maxThreadsPerBlock;
    // if (max_block_size % warp_size)
    //     // handle non-sensical return values from hipFuncGetAttributes
    //     max_block_size = (max_block_size / warp_size - 1) * warp_size;
    //
    // unsigned int run_block_size = min(block_size, max_block_size);
    //
    // // setup the grid to run the kernel
    // dim3 grid(N / run_block_size + 1, 1, 1);
    // dim3 threads(run_block_size, 1, 1);

    // run the kernel
    hipLaunchKernelGGL((gpu_compute_torsionaltrap_sin_force_kernel),
                       dim3(grid),dim3(threads),0,0,group_size,box,
                       d_pos,
                       d_torque,
                       d_rtag,
                       d_tag_array1,
                       d_tag_array2,
                       d_tag_array3,
                       d_tag_array4,
                       d_ref_angles,
                       d_angles,
                       d_oldnew_angles,
                       d_ref_vecp,
                       d_ref_vecn,
                       d_oldnew_value,
                       d_ref_vecp_value,
                       d_ref_vecn_value,
                       d_group_typeval,
                       d_params,
                       timestep);


    return hipSuccess;
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
