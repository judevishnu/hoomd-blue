#include "hip/hip_runtime.h"
// Copyright (c) 2009-2022 The Regents of the University of Michigan.
// Part of HOOMD-blue, released under the BSD 3-Clause License.

#include "TorsionalForceGPU.cuh"
#include "hoomd/TextureTools.h"
#include <iostream>
#include <math.h>
#include <sstream>
#include <stdexcept>
#include <vector>

#include <assert.h>

#ifdef SINGLE_PRECISION
#define __scalar2int_rn __float2int_rn
#else
#define __scalar2int_rn __double2int_rn
#endif

/*! \file HarmonicDihedralForceGPU.cu
    \brief Defines GPU kernel code for calculating the harmonic dihedral forces. Used by
   HarmonicDihedralForceComputeGPU.
*/

namespace hoomd
    {
namespace md
    {
namespace kernel
    {

//! GPU implementation of anglDiff
__device__ Scalar gpu_anglDiff(Scalar diff)
    {
    if (diff > M_PI)
        {
        diff -= 2 * M_PI;
        }
    else if (diff <= -M_PI)
        {
        diff += 2 * M_PI;
        }
    return diff;
    }
//! Kernel for calculating harmonic dihedral forces on the GPU
/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos particle positions on the device
    \param d_params Parameters for the angle force
    \param box Box dimensions for periodic boundary condition handling
    \param tlist Dihedral data to use in calculating the forces
    \param dihedral_ABCD List of relative atom positions in the dihedrals
    \param pitch Pitch of 2D dihedral list
    \param n_dihedrals_list List of numbers of dihedrals per atom
*/
__global__ void gpu_compute_torsional_sin_force_kernel(const unsigned int group_size,const BoxDim box,
                                                       const Scalar4* d_pos,
                                                       Scalar4* d_torque,
                                                       unsigned int* rtag,
                                                       unsigned int* d_index_array1,
                                                       unsigned int* d_index_array2,
                                                       unsigned int* d_index_array3,
                                                       unsigned int* d_index_array4,
                                                       Scalar* d_ref_angles,
                                                       Scalar* d_angles,
                                                       Scalar2* d_oldnew_angles,
                                                       const Index2D d_oldnew_value,
                                                       const typeval_union* d_group_typeval,
                                                       const Scalar4* d_params,
                                                       long unsigned int timestep)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= group_size)
        return;
    unsigned int typval = d_group_typeval[group_idx].type;
    //printf("%u \n", typva);
    Scalar4 params = __ldg(d_params + typval);
    //printf("%u %u %f %f %f %f \n", group_idx,typval,params.x,params.y,params.z,params.w);

    Scalar K = params.x;
    Scalar tqx = params.y;
    Scalar tqy = params.z;
    Scalar tqz = params.w;

    unsigned int tagp = d_index_array1[group_idx];
    unsigned int tagn = d_index_array2[group_idx];
    unsigned int tagpside = d_index_array3[group_idx];
    unsigned int tagnside = d_index_array4[group_idx];
    unsigned int rtagp = d_r_tag[tagp];
    unsigned int rtagn = r_tag[tagn];
    unsigned int rtagpside = d_r_tag[tagpside];
    unsigned int rtagnside = d_r_tag[tagnside];


    Scalar4 pos_b = __ldg(d_pos + tagp);
    Scalar4 pos_c = __ldg(d_pos + tagn);
    Scalar4 pos_a = __ldg(d_pos + tagpside);
    Scalar4 pos_d = __ldg(d_pos + tagnside);

    // Scalar4 pos_b = d_pos[tagp];
    // Scalar4 pos_c = d_pos[tagn];
    // Scalar4 pos_a = d_pos[tagpside];
    // Scalar4 pos_d = d_pos[tagnside];

    Scalar3 a_poss = make_scalar3(pos_a.x,pos_a.y,pos_a.z);
    Scalar3 b_poss = make_scalar3(pos_b.x,pos_b.y,pos_b.z);
    Scalar3 c_poss = make_scalar3(pos_c.x,pos_c.y,pos_c.z);
    Scalar3 d_poss = make_scalar3(pos_d.x,pos_d.y,pos_d.z);


    Scalar3 dab;
    dab = a_poss - b_poss;


    // dab.x = pos_a.x - pos_b.x;
    // dab.y = pos_a.y - pos_b.y;
    // dab.z = pos_a.z - pos_b.z;

    Scalar3 ddc;
    // ddc.x = pos_d.x - pos_c.x;
    // ddc.y = pos_d.y - pos_c.y;
    // ddc.z = pos_d.z - pos_c.z;

    ddc = d_poss - c_poss;


    dab = box.minImage(dab);

    ddc = box.minImage(ddc);

    //####################################################################################################
    Scalar angl;
    Scalar diffangl;
    Scalar ref_angl;
    Scalar tmpangl;
    Scalar oldangl;
    Scalar3 torqp;
    Scalar3 torqn;
    Scalar3 constT;
    torqp = make_scalar3(0,0,0);
    torqn = make_scalar3(0,0,0);
    tmpangl = 0;
    angl = 0;
    diffangl=0;
    ref_angl=0;

    tmpangl = atan2(dab.y, dab.x) - atan2(ddc.y, ddc.x);
    //tmpangl = gpu_anglDiff(tmpangl);
    if (tmpangl > M_PI)
        {
        tmpangl -= 2 * M_PI;
        }
    else if (tmpangl <= -M_PI)
        {
        tmpangl += 2 * M_PI;
        }
    oldangl = d_oldnew_angles[d_oldnew_value(group_idx, typval)].x;
    diffangl = tmpangl - oldangl;
    //diffangl = gpu_anglDiff(diffangl);
    if (diffangl > M_PI)
        {
        diffangl -= 2 * M_PI;
        }
    else if (diffangl <= -M_PI)
        {
        diffangl += 2 * M_PI;
        }
    d_oldnew_angles[d_oldnew_value(group_idx, typval)].y = tmpangl;
    angl = d_angles[group_idx]+diffangl;
    //printf("%d %f \n",i,angl);
    d_angles[group_idx] = angl;
    Scalar cs = slow::cos(angl);
    Scalar ss = slow::sin(angl);
    d_oldnew_angles[d_oldnew_value(group_idx, typval)].x = tmpangl;
    // Scalar distone = sqrt(dab.x*dab.x+dab.y*dab.y+dab.z*dab.z);
    // Scalar disttwo = sqrt(ddc.x*ddc.x+ddc.y*ddc.y+ddc.z*ddc.z);
    Scalar distone =  slow::rsqrt(dot(dab, dab));
    Scalar disttwo =  slow::rsqrt(dot(ddc, ddc));
    if(group_idx==0)
      {
      //printf("%u %u %u %u %u %u %f %f %f %f %f %f\n",timestep,group_idx,tagp,tagn,tagpside,tagnside,diffangl,tmpangl,oldangl,angl,distone,disttwo);
      printf("GPU %lu %u %u %u %u %u \n",timestep,group_idx,tagp,tagn,tagpside,tagnside);
      printf("GPU %lu %u %u %u %u %u \n",timestep,group_idx,rtagp,rtagn,rtagpside,rtagnside);


      }
    if ((angl> M_PI)&&(angl<3*M_PI/2))
      {
      ss = slow::sin(angl- M_PI);
      cs = slow::cos(angl- M_PI);
      torqp = make_scalar3(0,0,-2*K*cs*ss);
      torqn = make_scalar3(0,0,2*K*cs*ss);
      }
    else if (angl < 0)
      {
      Scalar cs = slow::cos(angl);
      Scalar ss = slow::sin(angl);
      torqp = make_scalar3(0,0,-2*K*cs*ss);
      torqn = make_scalar3(0,0,2*K*cs*ss);
      }
    // else if (angl == 0)
    //   {
    //   if (timestep < 10)
    //     {
    //     torqp.x =  tqx;
    //     torqp.y =  tqy;
    //     torqp.z =  tqz;
    //     torqn.x =  tqx;
    //     torqn.y =  tqy;
    //     torqn.z = -tqz;
    //     }
    //   }
    d_torque[tagp] = make_scalar4(torqp.x,torqp.y,torqp.z,0);
    d_torque[tagn] = make_scalar4(torqn.x,torqn.y,torqn.z,0);
    }

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos particle positions on the GPU
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param tlist Dihedral data to use in calculating the forces
    \param dihedral_ABCD List of relative atom positions in the dihedrals
    \param pitch Pitch of 2D dihedral list
    \param n_dihedrals_list List of numbers of dihedrals per atom
    \param d_params K, sign,multiplicity params packed as padded Scalar4 variables
    \param n_dihedral_types Number of dihedral types in d_params
    \param block_size Block size to use when performing calculations
    \param compute_capability Compute capability of the device (200, 300, 350, ...)

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one Scalar4 element per dihedral type. The x component contains K the
   spring constant and the y component contains sign, and the z component the multiplicity.
*/
hipError_t gpu_compute_torsional_sin_forces(const unsigned int group_size,const BoxDim& box,
                                                const Scalar4* d_pos,
                                                Scalar4* d_torque,
                                                unsigned int* rtag,
                                                unsigned int* d_index_array1,
                                                unsigned int* d_index_array2,
                                                unsigned int* d_index_array3,
                                                unsigned int* d_index_array4,
                                                Scalar* d_ref_angles,
                                                Scalar* d_angles,
                                                Scalar2* d_oldnew_angles,
                                                const Index2D& d_oldnew_value,
                                                const typeval_union* d_group_typeval,
                                                // const size_t virial_pitch,
                                                // const unsigned int N,
                                                // const Scalar4* d_pos,
                                                // const BoxDim& box,
                                                // const group_storage<4>* tlist,
                                                // const unsigned int* dihedral_ABCD,
                                                // const unsigned int pitch,
                                                // const unsigned int* n_dihedrals_list,
                                                Scalar4* d_params,
                                                unsigned int timestep,
                                                // unsigned int n_dihedral_types,
                                                long unsigned int block_size)
                                                //,int warp_size)
    {
    assert(d_params);
    // setup the grid to run the kernel
    dim3 grid(group_size / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);
    // unsigned int max_block_size;
    // hipFuncAttributes attr;
    // hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_compute_harmonic_dihedral_forces_kernel);
    // max_block_size = attr.maxThreadsPerBlock;
    // if (max_block_size % warp_size)
    //     // handle non-sensical return values from hipFuncGetAttributes
    //     max_block_size = (max_block_size / warp_size - 1) * warp_size;
    //
    // unsigned int run_block_size = min(block_size, max_block_size);
    //
    // // setup the grid to run the kernel
    // dim3 grid(N / run_block_size + 1, 1, 1);
    // dim3 threads(run_block_size, 1, 1);

    // run the kernel
    hipLaunchKernelGGL((gpu_compute_torsional_sin_force_kernel),
                       dim3(grid),dim3(threads),0,0,group_size,box,
                       d_pos,
                       d_torque,
                       rtag,
                       d_index_array1,
                       d_index_array2,
                       d_index_array3,
                       d_index_array4,
                       d_ref_angles,
                       d_angles,
                       d_oldnew_angles,
                       d_oldnew_value,
                       d_group_typeval,
                       d_params,
                       timestep);
                       // 0,
                       // 0,
                       // d_force,
                       // d_virial,
                       // virial_pitch,
                       // N,
                       // d_pos,
                       // d_params,
                       // box,
                       // tlist,
                       // dihedral_ABCD,
                       // pitch,
                       // n_dihedrals_list);

    return hipSuccess;
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
